// cublas_gemm.cu
#include <cstdio>
#include <cstdlib>
#include <cmath>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

// Matrix size
static const int N = 1024;

//------------------------------------------------------------------------------
// CPU reference multiply
void cpuReferenceGemmHalf(const half* A, const half* B, float* C_ref, int N) {
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++){
            float sum = 0.0f;
            for(int k = 0; k < N; k++){
                float a_ij = __half2float(A[i*N + k]);
                float b_jk = __half2float(B[k*N + j]);
                sum += a_ij * b_jk;
            }
            C_ref[i*N + j] = sum;
        }
    }
}

//------------------------------------------------------------------------------
// Compare GPU result (in half) to CPU reference (in float)
bool compareResultsHalfToFloat(const half* GPU_result, const float* CPU_result, 
                               int N, float tolerance = 1e-3f) {
    for(int i = 0; i < N*N; i++){
        float gpu_val = __half2float(GPU_result[i]);
        float cpu_val = CPU_result[i];
        float diff = fabs(gpu_val - cpu_val);
        float relative = diff / (fabs(cpu_val) + 1e-7f);

        if (diff > tolerance && relative > tolerance) {
            printf("Mismatch at index %d: GPU=%f, CPU=%f, diff=%f\n", 
                   i, gpu_val, cpu_val, diff);
            return false;
        }
    }
    return true;
}

//------------------------------------------------------------------------------
int main()
{
    size_t size = N * N * sizeof(half);
    half* h_A = (half*)malloc(size);
    half* h_B = (half*)malloc(size);
    half* h_C = (half*)malloc(size);

    // CPU ref in float
    float* h_Ref = (float*)malloc(N * N * sizeof(float));

    // Init host data
    for (int i = 0; i < N*N; i++){
        float valA = static_cast<float>(rand() % 3);
        float valB = static_cast<float>(rand() % 3);
        h_A[i] = __float2half(valA);
        h_B[i] = __float2half(valB);
    }

    // CPU reference
    printf("Running CPU reference multiply ...\n");
    cpuReferenceGemmHalf(h_A, h_B, h_Ref, N);

    // Device memory
    half *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // cuBLAS handle
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // Enable Tensor Core math
    hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH);

    float alpha = 1.0f;
    float beta  = 0.0f;

    // N x N = (N x N) * (N x N)
    // C = A * B in half precision, with float alpha/beta
    hipblasStatus_t stat = hipblasGemmEx(
        handle,
        HIPBLAS_OP_N, HIPBLAS_OP_N,
        N, N, N,
        &alpha,
        d_A, HIP_R_16F, N,
        d_B, HIP_R_16F, N,
        &beta,
        d_C, HIP_R_16F, N,
        HIP_R_16F,
        CUBLAS_GEMM_DEFAULT_TENSOR_OP
    );

    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf("hipblasGemmEx failed\n");
    }

    // Copy result back
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Check results
    printf("Comparing GPU result to CPU reference...\n");
    bool pass = compareResultsHalfToFloat(h_C, h_Ref, N);
    if (!pass) {
        printf("ERROR: Results do not match!\n");
    } else {
        printf("PASS: GPU results match CPU reference.\n");
    }

    // Cleanup
    hipblasDestroy(handle);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);
    free(h_Ref);

    printf("cuBLAS GEMM done.\n");
    return 0;
}

